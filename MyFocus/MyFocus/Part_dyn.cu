#include "hip/hip_runtime.h"
/*
Dinámica de partículas cargadas en tokamaks con CUDA
Adaptado del código FOCUS -> https://doi.org/10.1016/j.cpc.2018.07.0180010-4655
Facundo Sheffield - 2022
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "Random123/philox.h"  // random numbers in GPU
#include "Random123/u01.h"  // to get uniform deviates [0,1]


#include "curso.h"


/* ********** Global parameters. ******************/
// Plasma and reactor parameters  (normalized, DIIID)
// HOST ******************
const double hB_0   = 2.2;				// Campo Magnetico Toroidal en el eje (Tesla)
const double a   = 0.67 / 1.67;			// Radio menor del Toroide (previously a_cm)
const double pitch_deg = 60; 			// Pitch en grados, para Init_CI_costado
const int gridsize = 10;					// Gridsize, para Init_CI_costado
const double delta = 0.61;				// equilibrium triangularity
const double R   = 1;				// Radio mayor del Toroide (previously R_cm)
const double hR0    = R;                       // Radio normalizado
const double Ep_MeV = .08;                              // Energía del proyectil (inicial, Mev)
const double hmu    = 2.0;                              // fracción masa proyectil/masa proton (creo)
const int    Npart  = gridsize*gridsize;           // Numero de partículas
const int    hNstep = 56000000;				// Limite paso temporales.
const int m_steps = 10; 					// number of time steps to measure position
const double hDt    = 0.16;                              // Temporal step (normalized)
const double hZp    = 1.0;                              // Numero atomico proyectil
double hgamma;
const double hta    = 1.0439E-8*hmu/(hZp*hB_0);		// sec. (Campo en Teslas) (ITER: 3.94E-9 s) (PREGUNTAR-> el tiempo de ciclotron)
// constants (Used for adimensionalizations)
const double hc_cgs = 2.9979E10;			// speed of light
const double ha0_cgs = 5.2918E-9;			// Bohr radius
const double hmp_au = 1836.2;				// proton mass (in a.u.)

// DEVICE ************************
__device__ double dEp_MeV= Ep_MeV;
__device__ double E_0    = 0.0;				// Campo electrico de referencia
__device__ double B_0    = hB_0;
__device__ double R0     = R;
__device__ double mu_i   = 1.0;				// Fraccion masa ion-target/masa proton. (PREGUNTAR, colisiones?)
__device__ double Zb     = 1.0;				// Atomic number of Plasma ions 
__device__ double n_e    = 1.0;				// Densidad e- y p+ del Plasma (CORE) (10E14 cm-3)
__device__ double nH     = 0.01;			// Hydrogen impurities (10E14 cm-3)
__device__ double nH2    = 0.005;			// Molecular H impurity (10E14 cm-3)
__device__ double nHe    = 0.005;			// Idem Helio.
__device__ int    Nstep  = hNstep;			// Limite pasos temporales
//__device__ int    Nec	 = 100;			// Cada cuanto computo las colisiones elásticas.
__device__ double Zp     = hZp;				// Numero atomico proyectil
__device__ double mu     = hmu;             // fraccion masa proyectil/masa proton
__device__ double Dt     = hDt;				//
__device__ double da_cm  = a;
__device__ double ta     = hta;
__device__ __constant__ double PI = 3.1415926535897932385;
__device__ double c_cgs = hc_cgs;
__device__ double a0_cgs = ha0_cgs;
__device__ double mp_au = hmp_au;

 struct Part {
	double E_keV; 			// Energia en keV. (los files estan en esta unidad).
	int Z;  				// Numero atomico.
	int q;					// carga neta. (Adimensional)		
	double r[3];			// posición. Coord. cilindricas (r, theta, z). (Adimensional)
	double v[3];			// velocidad. Coord. cilindricas. (Adimensional)
	double time;			// time of particle evolution.
	int state;				// -1 = sin determinar; 0 = escapada; 1 = banana; 2 = clockwise; 3 = anticlockwise; 4 = outlier
	int sense;				// sense of rotation
    double pitch;  			// Vparalela al campo (V_par/V=cos(pitch))
    double flux;
    int flag; 				// Indica algún flag, en este caso es 1 si salió y volvió a entrar y 0 else
	};

#include "Magnetic_field.h"
#include "General_functions.h" // Utiliza "struct Part" y los #define a_cm, mu, etc!! -> Por eso el include está luego de struct Part
#include "Elastic_collision_module.h"  // Módulo de colisiones elásticas

// Control Trayectoria:
struct Position {
	double r[3];
	double rg[3];
	};
// ----------------
//da la proyección (v_paralela) y actualiza el valor del flujo. Es un poco confuso
__host__ __device__ double Proyection(double r,double z, double vr,double vt,double vz,double *s_flux){
	//variables para las velocidades
	double v[3]={0.0};
	double psi;
	//Variables para los campos para no calcular repetidamente
	double B_equilibrio[3], modB=0.0;	
	//variables auxiliares
	double proyection=0.0;
	double qq,time,y;
	//empiezo el calculo de las velocidades
	// B_Asdex(r,z, &B_equilibrio[0], &psi);		 
	// B_Asdex(B, E,r, qq, z,time,y);
	B_Analitico(r,z, &B_equilibrio[0], &psi);  // actualiza el valor de B y s_flux con el eq. analitico
	*s_flux=psi;

	double	Br=B_equilibrio[0];
	double	Bt=B_equilibrio[1];				
	double	Bz=B_equilibrio[2];				
		
		modB=sqrt( Br*Br + Bt*Bt + Bz*Bz );
		//versor paralelo a B
		v[0]=Br/modB;
		v[1]=Bt/modB;
		v[2]=Bz/modB;

		proyection = vr*v[0] + vt*v[1] + vz*v[2];
		//printf("proyection= %e\n", proyection);
		return proyection;				
		}

//Control Trayectoria/Evolución temporal: ---------------------------

__global__ void Evolution ( struct Part * d_He, int Npart, long init) {
	//Evolución temporal "normal", asigna los tipos de órbitas en d_He.state

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int Nec;		//steps for elastic collisions
	int Nic = 1000; 	//steps for inelastic collisions
	int n = 0;
	int i,kk;	//semilla random numbers
	int q1 = 2;
	double qq00,qq11,omega=16.e-4,tiempo,tiempo0; //flag
	double s_flux;

	double y;// gamma

	//sense variables -----------
	double t0 = 0.0;
	double vpar, vpar_t;
	double B[3], E[3];
	y = sqrt(10*Ep_MeV) * 0.01758437;  // gamma DIIID, 0.0175 es para 100 keV
	double initial_proyection=0.0,proyection=0.0;
	//Control Trayectoria: ------- 
	int j = 0;

	kk=0;
	short unsigned int Period_tol = 40;  // N° de pasos temporales que puede estar fuera del eq (~1 períodos)

	// Colisiones elásticas:

	// Random numbers C :
	// Random numbers initialization -------------
	philox2x32_ctr_t   c={{}}; 
	philox2x32_ukey_t  uk = {{}}; 
	uk.v[0] = id + (int)init;		 
	philox2x32_key_t   k = philox2x32keyinit(uk); 
	philox2x32_ctr_t   p;	 
	double Ran1,Ran2; 
	double Ran_EC[4];    // pitch-energy 
	//double Ran_EC[6];  // euler	

	for(i=0;i<1000;i++){ 
	 	c.v[0] = i; 
	 	p = philox2x32(c, k); 
	} 


	if(id < Npart) {
		n = 0; 		
		
		// ya están en las condiciones iniciales	
		//initial_proyection = Proyection(d_He[id].r[0],d_He[id].r[2],d_He[id].v[0],d_He[id].v[1],d_He[id].v[2],&s_flux);
		//d_He[id].flux=s_flux;

		initial_proyection = d_He[id].pitch;
		//printf("proyection= %e\n", initial_proyection);  //proyection= 9.000147e-01 para debuggear

		short unsigned int out_counter = 0;  // if out_counter = Period_tol -> escapada
		bool is_in = true;
		bool was_outside = false;


		do{ 

			RK46_NL(d_He+id, y);
			//Boris_c(d_He+id, y);
			n++;

			// Elastic collisions: ------------------------------------------
			if(d_He[id].E_keV > 700){
				Nec = 500000;
			}else if(d_He[id].E_keV <= 700 && d_He[id].E_keV > 310){
				Nec = 200000;
			}else{
				Nec = 100000;
			}
			if(n % Nec == 0){
				Elastic_collisions(d_He+id, 1.0*Nec*Dt, &i, init, id);
				// Randoms numbers needed in elast. collision -------
				/*
				i = i + 1;
				c.v[0] = i;		// Some loop-dependent application
				i = i + 1;
				c.v[1] = i;		// another loop-dependent application variable.
				p = philox2x32(c, k);
				Ran_EC[0] = (double) u01_open_open_32_53(p[0]);
				Ran_EC[1] = (double) u01_open_open_32_53(p[1]);
				//Ran_gauss(&Ran_EC[0]); // Activar para euler
				i = i + 1;
				c.v[0] = i;		
				i = i + 1;
				c.v[1] = i;	
				p = philox2x32(c, k);
				Ran_EC[2] = (double) u01_open_open_32_53(p[0]);
				Ran_EC[3] = (double) u01_open_open_32_53(p[1]);
				Ran_gauss(&Ran_EC[2]);
				i = i + 1;
				c.v[0] = i;		
				i = i + 1;
				c.v[1] = i;	
				p = philox2x32(c, k);
				Ran_EC[4] = (double) u01_open_open_32_53(p[0]);
				Ran_EC[5] = (double) u01_open_open_32_53(p[1]);
				Ran_gauss(&Ran_EC[4]);
				*/

				// __device__ void Elastic_collisions_SV_euler (struct Part *He, double Dt, double * Ran_EC);
				//Elastic_collisions_PE_MC(d_He+id,(double)Nec*Dt, &Ran_EC[0]);
				//Elastic_collisions_euler (d_He+id,(double)Nec*Dt, &Ran_EC[0]);
				// (Notar que para euler necesito 6 nros gaussianos!!!!)
			}

			// ----------------------------------------------------------------
			// nueva vparalela
			proyection=Proyection(d_He[id].r[0],d_He[id].r[2],d_He[id].v[0],d_He[id].v[1],d_He[id].v[2],&s_flux);
			/*if(n==3){
				printf("proyection= %e\n", proyection);  // for debug
			}*/
			d_He[id].flux=s_flux;
			if(s_flux<0){  
				was_outside = true;
				out_counter++;

				is_in = false;
                
				if (out_counter == Period_tol) {
					d_He[id].state=0;  // escapada
                    break;
                }
			} else {
				is_in = true;
				out_counter=0;
				if(was_outside && is_in){
					d_He[id].flag = 1;
				}
			}

			if(n>500000 && (proyection*initial_proyection)<0){  // puedo ponerle más condiciones para determinar mejor las órbitas
				d_He[id].state=1;  // banana
				// break;  // puedo comentar el break para ver la órbita completa
			}
			d_He[id].pitch=proyection;
			
		}while(n<Nstep && d_He[id].state != 0 );

		if (d_He[id].E_keV > 0){
			if (d_He[id].state == -1){  // sin asignar 
				if(d_He[id].pitch>0){
					d_He[id].state = 2;  // Clockwise
				} else if(d_He[id].pitch<0){
					d_He[id].state = 3;  // Anticlockwise
				} else {
					d_He[id].state = 4;  // Outlier!
				}
			} 
		} else {
			d_He[id].state = 4;  // Outlier, energía cinética negativa o NaN
		}
	}
}

// Evoluciona una particula (con CUDA) y retorna un puntero con las coordenadas en función del tiempo
__global__ void SingleEvol ( struct Part * d_He,  long init, int ip, struct Position * d_R) {
	//Evolución temporal "normal", asigna las orbitas en d_He.state
	int Npart = 1;
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int Nec;		//steps for elastic collisions
	int Nic = 1000; 	//steps for inelastic collisions
	int n = 0;
	int i,kk;	//semilla random numbers
	int q1 = 2;
	double qq00,qq11,omega=16.e-4,tiempo,tiempo0; //flag
	double s_flux;

	double y;// gamma

	//sense variables -----------
	double t0 = 0.0;
	double vpar, vpar_t;
	double B[3], E[3];
	y = sqrt(10*Ep_MeV) * 0.01758437;  // gamma DIIID, 0.0175 es para 100 keV
	double initial_proyection=0.0,proyection=0.0;
	//Control Trayectoria: ------- 
	int j = 0;

	kk=0;
	// Period tol no debería ser mayor a 6
	short unsigned int Period_tol = 40;  // N° de pasos temporales que puede estar fuera del eq (~1 períodos)

	// Colisiones elásticas:

	// Random numbers C :
	// Random numbers initialization -------------
	philox2x32_ctr_t   c={{}}; 
	philox2x32_ukey_t  uk = {{}}; 
	uk.v[0] = id + (int)init;		 
	philox2x32_key_t   k = philox2x32keyinit(uk); 
	philox2x32_ctr_t   p;	 
	double Ran1,Ran2; 
	double Ran_EC[4];    // pitch-energy 
	//double Ran_EC[6];  // euler	

	for(i=0;i<1000;i++){ 
	 	c.v[0] = i; 
	 	p = philox2x32(c, k); 
	} 

	if(id < Npart) {
		n = 0; 		
		int m = 0;
		// ya están en las condiciones iniciales	
		//initial_proyection = Proyection(d_He[id].r[0],d_He[id].r[2],d_He[id].v[0],d_He[id].v[1],d_He[id].v[2],&s_flux);
		//d_He[id].flux=s_flux;

		initial_proyection = d_He[id].pitch;
		//printf("proyection= %e\n", initial_proyection);  //proyection= 9.000147e-01 para debuggear

		short unsigned int out_counter = 0;  // if out_counter = Period_tol -> escapada
		bool is_in = true;
		bool was_outside = false;


		do{ 
			if(n % m_steps == 0){
				// guardando la posición (no queda lindo para m_steps > 5)
				// d_R[m].r[0] = d_He[0].r[0];	d_R[m].r[1] = d_He[0].r[1];	d_R[m].r[2] = d_He[0].r[2];

				// guardando el centro de giro:
				double cg[3];
				centro_giro(d_He+id, cg, y);  
				d_R[m].r[0] = cg[0];	d_R[m].r[1] = cg[1];	d_R[m].r[2] = cg[2];
				m++;
			}
			
			RK46_NL(d_He+id, y);
			//Boris_c(d_He+id, y);
			n++;

			// Elastic collisions: ------------------------------------------
			if(d_He[id].E_keV > 700){
				Nec = 500000;
			}else if(d_He[id].E_keV <= 700 && d_He[id].E_keV > 310){
				Nec = 200000;
			}else{
				Nec = 100000;
			}
			if(n%Nec == 0){
				// printf("Collision! \n");
				Elastic_collisions(d_He+id, 1.0*Nec*Dt, &i, init, id);

				// Randoms numbers needed in elast. collision -------
				/*
				i = i + 1;
				c.v[0] = i;		// Some loop-dependent application
				i = i + 1;
				c.v[1] = i;		// another loop-dependent application variable.
				p = philox2x32(c, k);
				Ran_EC[0] = (double) u01_open_open_32_53(p[0]);
				Ran_EC[1] = (double) u01_open_open_32_53(p[1]);
				//Ran_gauss(&Ran_EC[0]); // Activar para euler
				i = i + 1;
				c.v[0] = i;		
				i = i + 1;
				c.v[1] = i;	
				p = philox2x32(c, k);
				Ran_EC[2] = (double) u01_open_open_32_53(p[0]);
				Ran_EC[3] = (double) u01_open_open_32_53(p[1]);
				Ran_gauss(&Ran_EC[2], 1);  // mean, sd
				i = i + 1;
				c.v[0] = i;		
				i = i + 1;
				c.v[1] = i;	
				p = philox2x32(c, k);
				Ran_EC[4] = (double) u01_open_open_32_53(p[0]);
				Ran_EC[5] = (double) u01_open_open_32_53(p[1]);
				Ran_gauss(&Ran_EC[4], 1);
				*/	

				// __device__ void Elastic_collisions_SV_euler (struct Part *He, double Dt, double * Ran_EC);
				//Elastic_collisions_PE_MC_euler(d_He+id,(double)Nec*Dt, &Ran_EC[0]);
				//Elastic_collisions_SV_euler (d_He+id,(double)Nec*Dt, &Ran_EC[0]);
				//Elastic_collisions_euler (d_He+id,(double)Nec*Dt, &Ran_EC[0]);
				// (Notar que para euler necesito 6 nros gaussianos!!!!)
			}


			// ----------------------------------------------------------------
			// nueva vparalela
			proyection=Proyection(d_He[id].r[0],d_He[id].r[2],d_He[id].v[0],d_He[id].v[1],d_He[id].v[2],&s_flux);
			/*if(n==3){
				printf("proyection= %e\n", proyection);  // for debug
			}*/
			d_He[id].flux=s_flux;
			if(s_flux<0){  
				was_outside = true;
				out_counter++;

				is_in = false;
                
				if (out_counter == Period_tol) {
					d_He[id].state=0;  // escapada
                    break;
                }
			} else {
				is_in = true;
				out_counter=0;
				if(was_outside && is_in){
					d_He[id].flag = 1;
				}
			}

			if(n>400000 && (proyection*initial_proyection)<0){  // puedo ponerle más condiciones para determinar mejor las órbitas
				d_He[id].state=1;  // banana
				// break;  // puedo comentar el break para ver la órbita completa
			}
			d_He[id].pitch=proyection;
			
		}while(n<Nstep && d_He[id].state != 0 );

		if (d_He[id].E_keV > 0){
			if (d_He[id].state == -1){  // sin asignar 
				if(d_He[id].pitch>0){
					d_He[id].state = 2;  // Clockwise
				} else if(d_He[id].pitch<0){
					d_He[id].state = 3;  // Anticlockwise
				} else {
					d_He[id].state = 4;  // Outlier!
				}
			} 
		} else {
			d_He[id].state = 4;  // Outlier, energía cinética negativa o NaN
		}
	}
}



// Evoluciona una particula (sin CUDA) y genera un file con las coordenadas en función del tiempo (not working)
void singleP_Evol_Not_implemented (struct Part *d_He2, int ip){
		//# no será que hay problemas de nombre????
		FILE *File_orbit = fopen("singleP_Evol.dat","w");  // Creates a File
		if(File_orbit == NULL){
			printf("Error File_orbit");
			exit(1);}  
			
		fprintf(File_orbit,"# Particula N: %d\n",ip);

		int id = 0;
		
		int Nec;		//steps for elastic collisions
		int Nic = 1000; 	//steps for inelastic collisions
		int n = 0;
		int i,kk;	//semilla random numbers
		int q1 = 2;
		double qq00,qq11,omega=16.e-4,tiempo,tiempo0; //flag
		double s_flux;

		double y;  // gamma

		//sense variables ------------
		double t0 = 0.0;
		double vpar, vpar_t;
		double B[3], E[3];
		y = sqrt(10*Ep_MeV) * 0.01758437;  // gamma DIIID, 0.0175 es para 100 keV
		double initial_proyection=0.0,proyection=0.0;

		//Control Trayectoria: ------- 
		int j = 0;
		kk=0;
		short unsigned int Period_tol = 20;  // N° de pasos temporales que puede estar fuera del eq (1.5 períodos)
		int Allowed_Nstep = hNstep;  // en caso de que salga de del equilibrio

		

			n = 0; 		
			initial_proyection = (*d_He2).pitch;  
			//printf("proyection= %e\n", initial_proyection);  //proyection= 9.000147e-01 para debuggear

			short unsigned int out_counter = 0;  // if out_counter = Period_tol -> escapada
			bool is_in = true;
			bool was_outside = false;

			printf("%f \t %f \t %f \n",(*d_He2).r[0], (*d_He2).r[1], (*d_He2).r[2]);
			do{ 
				H_RK46_NL(d_He2, y);
				//Boris_c(d_He+id, y);

				// saves the full trajectory
				/*if(n==0){
					printf("%f \t %f \t %f \n",(*d_He2).r[0], (*d_He2).r[1], (*d_He2).r[2]);
				}
				fprintf(File_orbit,"%f \t %f \t %f \n",(*d_He2).r[0], (*d_He2).r[1], (*d_He2).r[2]);
				*/
				n++;
				// ----------------------------------------------------------------
				// nueva vparalela
				proyection=Proyection((*d_He2).r[0],(*d_He2).r[2],(*d_He2).v[0],(*d_He2).v[1],(*d_He2).v[2],&s_flux);
				/*if(n==3){
					printf("proyection= %e\n", proyection);  // for debug
				}*/
				(*d_He2).flux=s_flux;
				if(s_flux<0){  
					was_outside = true;
					out_counter++;

					if (n<hNstep-Period_tol && is_in){
						Allowed_Nstep = n + Period_tol;
					}

					is_in = false;
					
					if (out_counter == Period_tol) {
						(*d_He2).state=0;  // escapada
						break;
					}
				} else {
					is_in = true;
					out_counter=0;
					if(was_outside && is_in){
						(*d_He2).flag = 1;
					}
				}

				if(n>10000 && (proyection*initial_proyection)<0){  // puedo ponerle más condiciones para determinar mejor las órbitas
					(*d_He2).state=1;  // banana
					break;
				}
				(*d_He2).pitch=proyection;
				
			}while(n<Allowed_Nstep && (*d_He2).state != 0 );

			if ((*d_He2).state == -1){  // sin asignar 
				if((*d_He2).pitch>0){
					(*d_He2).state = 2;  // Clockwise
				} else if((*d_He2).pitch<0){
					(*d_He2).state = 3;  // Anticlockwise
				} else {
					(*d_He2).state = 4;  // Outlier!
				}
			}
		fclose(File_orbit);
}

int main(){
	// Nota: en realidad xx = r, yy=theta, zz=z
    double xx[Npart],yy[Npart],zz[Npart],vx[Npart],vy[Npart],vz[Npart],tiempo,s_flux;
	
	struct Part He[Npart];
	int ip;         			// Particle index
	double x,r;				// Initial x position (initializated in fn init_r)
	double rg[3];				// Guiding center.

	int Part_charge[3]={0,0,0};		// Final charge state counter.
	
	struct timeval start;			// Computational time.
	struct timeval finish; 
	double elapsed_time;
	
	/* *********** Output files  ***************/
	FILE *File_IC = fopen("time0000.dat","w");  // Initial Conditions
	if(File_IC == NULL){
		printf("Error File_IC");
		exit(1);}

	FILE *File_FC = fopen("time0001.dat","w");  // Final Conditions
	if(File_FC == NULL){
		printf("Error File_FC");
		exit(1);}

	FILE *File_St = fopen("SR_1MeV0_0x20_He_e_euler.dat","w");  // stats
	if(File_St == NULL){
		printf("Error File_St");
		exit(1);}

	FILE *File_Orbit_types = fopen("Orbits.dat","w");  // Conteo de órbitas
	if(File_Orbit_types == NULL){
		printf("Error File_Orbit_types");
		exit(1);}

	/*********************************************/ 

	/* Random numbers initialization *****/
	double f;
	time_t tran = time(NULL);
	init = labs(init - tran);
	//init = labs(init - 10);  //same velocity for each particle on every iteration
	long *ptrinit = &init;
	for(ip=0;ip<1000;ip++)
		f = ran2(ptrinit);		

	/* ***** Particle Initialization *********/

	//Init_rv(&xx[0],&yy[0],&zz[0],&vx[0],&vy[0],&vz[0],&tiempo,Npart);
	Init_CI_costado(&xx[0],&yy[0],&zz[0],&vx[0],&vy[0],&vz[0], pitch_deg, gridsize, delta);

	double hgamma = sqrt(10 * Ep_MeV) * 0.01758437;

	for(ip=0;ip<Npart;ip++){
		He[ip].E_keV = Ep_MeV*1000.0;
		He[ip].Z = (int)hZp;
		He[ip].q = (int)hZp;

		He[ip].r[0]=xx[ip];
		He[ip].r[1]=yy[ip];
		He[ip].r[2]=zz[ip];
		He[ip].v[0]=vx[ip];
		He[ip].v[1]=vy[ip];
		He[ip].v[2]=vz[ip];

		// v_paralela y flujo 
		He[ip].pitch = Proyection(He[ip].r[0],He[ip].r[2],He[ip].v[0],He[ip].v[1],He[ip].v[2],&s_flux);
		He[ip].flux = s_flux;
		He[ip].flag = 0;
		
		if(He[ip].pitch>0)
		  He[ip].sense =1;
		else
		  He[ip].sense =-1;
		
		if(s_flux<0)		  
		  He[ip].state = 0;
		else
		  He[ip].state = -1;
		
		He[ip].time = 0.0;
		fprintf(File_IC,"Número - tiempo - r - theta - z - Vr - Vtheta - Vz - E (kev) - psi - vparalela - sentido\n");
		fprintf(File_IC,"%d %f %f \t %f \t %f \t %f \t %f \t %f %f %f %f %d\n",ip,He[ip].time, He[ip].r[0], He[ip].r[1],He[ip].r[2],
			He[ip].v[0], He[ip].v[1], He[ip].v[2],He[ip].E_keV,He[ip].flux,He[ip].pitch, He[ip].sense);

	}

	printf("E in: %.14f keV \n", He[0].E_keV);
	printf("x in: %.14f \n", x);

	
	/* ***** Output Statistical */
	fprintf(File_St, "gamma: \t %f \n", sqrt(10*Ep_MeV) * 0.01758437);
	fprintf(File_St,"Initial \n");
	fprintf(File_St, "Dt: \t %f \n", hDt);
	fprintf(File_St, "Nstep: \t %d \n", hNstep);
	fprintf(File_St, "Simul time: \t %f msec. \n", (double)hNstep*hDt*hta*1000.0);
	fprintf(File_St, "Npart: \t %d \n", Npart);
	fprintf(File_St, "Z beam: \t %d \n", (int)hZp);
	fprintf(File_St, "Ep: \t %f keV \n", Ep_MeV*1000.0);
	fprintf(File_St, "x: \t %f \n", x);


	/* ******** Particle evolution **********/
	
	gettimeofday(&start,NULL);

	/***** CUDA ******/
	struct Part *d_He;
	HANDLE_ERROR(hipMalloc( (void**) &d_He, Npart*sizeof(Part) ));
    HANDLE_ERROR(hipMemcpy( d_He, &He, Npart*sizeof(Part), hipMemcpyHostToDevice ));
	checkCUDAError("Particle copy: failed \n");
	
    int dev = 0;
        if(hipGetDevice(&dev)!= hipSuccess)
                printf("hipGetDeviceCount FAILED");

        hipDeviceProp_t deviceProp;
        //for(int dev = 0; dev < deviceCount; ++dev){
                hipGetDeviceProperties(&deviceProp,dev);
                printf("\nPlaca %d: %s \n", dev, deviceProp.name);
	int numthreads = 32;  // 32, se puede probar con otras potencias de 2 para optimizar
	int numblocks = (Npart+numthreads-1)/numthreads;
	dim3 block_size(numthreads);
  	dim3 grid_size(numblocks);

	//Control Trayectoria: -------------------------
	Evolution<<< grid_size,block_size >>> (d_He, Npart, init);
	

	checkCUDAError("Kernel GPU: failed \n");
	/*  ********   */
	HANDLE_ERROR(hipMemcpy(&He, d_He, Npart*sizeof(Part), hipMemcpyDeviceToHost));
	checkCUDAError("copy to CPU: failed \n");
	HANDLE_ERROR(hipFree(d_He));

	int jj;

	// Posiciones finales y estadisticas--------------------
	fprintf(File_FC,"Número - tiempo - r - theta - z - Vr - Vtheta - Vz - E (kev) - psi - pitch - sentido\n");
	fprintf(File_Orbit_types, "# Particle Trajectories statistics, pitch=%f, delta=%f\n", pitch_deg, delta);
	fprintf(File_Orbit_types, "# Escapadas\tBananas\tClockwise\tAnticlockwise\tOutliers\n");
	
	int bananas=0; int clockW = 0; int anticlockW = 0; int escapadas = 0; int Outliers = 0;
	int reentrantes = 0;
	
	bool only_oneP = true;
	for(ip=0;ip<Npart;ip++){
		
		if (only_oneP && He[ip].state == 4){
			only_oneP = false;
			printf("Particle state: %d", He[ip].state);
			printf("\nFlag Particle, ip=%d\n", ip);
			// reseteo el estado de la partícula ip
			He[ip].E_keV = Ep_MeV*1000.0;
			He[ip].Z = (int)hZp;
			He[ip].q = (int)hZp;

			He[ip].r[0]=xx[ip];
			He[ip].r[1]=yy[ip];
			He[ip].r[2]=zz[ip];
			He[ip].v[0]=vx[ip];
			He[ip].v[1]=vy[ip];
			He[ip].v[2]=vz[ip];

			printf("Coordenada inicial radial r=%f\n", He[ip].r[0]);
			printf("E (keV): %f", He[ip].E_keV);

			// v_paralela y flujo 
			He[ip].pitch = Proyection(He[ip].r[0],He[ip].r[2],He[ip].v[0],He[ip].v[1],He[ip].v[2],&s_flux);
			He[ip].flux = s_flux;
			He[ip].flag = 0;
			
			if(He[ip].pitch>0){
			He[ip].sense = 1;}
			else{
			He[ip].sense = -1;}
			
			if(s_flux<0){		  
			He[ip].state = 0;}
			else{
			He[ip].state = -1;}
			
			He[ip].time = 0.0;

			int R_size = hNstep/m_steps;
			struct Position R[R_size];
			struct Position *d_R;
			struct Part *D_HE;

			hipMalloc( (void**) &d_R, R_size*sizeof(Position) );
			hipMemcpy( d_R, &R, R_size*sizeof(Position), hipMemcpyHostToDevice );
			checkCUDAError("Trajectory copy: failed \n");
			
			hipMalloc( (void**) &D_HE, 1*sizeof(Part) );
			hipMemcpy( D_HE, &He[ip], 1*sizeof(Part), hipMemcpyHostToDevice );
			checkCUDAError("Particle copy: failed \n");

			int dev = 0;
			if(hipGetDevice(&dev)!= hipSuccess)
					printf("hipGetDeviceCount FAILED");

			hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp,dev);
            printf("\nPlaca %d: %s \n", dev, deviceProp.name);
			int numthreads = 32;
			int numblocks = (Npart+numthreads-1)/numthreads;
			dim3 block_size(numthreads);
			dim3 grid_size(numblocks);

			SingleEvol<<< grid_size,block_size >>>  (D_HE, init, ip, d_R);

			checkCUDAError("Kernel GPU: failed \n");
			HANDLE_ERROR(hipFree(D_HE));

			HANDLE_ERROR(hipMemcpy( &R, d_R, R_size*sizeof(Position), hipMemcpyDeviceToHost ));
			checkCUDAError("copy to CPU R: failed \n");
			HANDLE_ERROR(hipFree(d_R));

			FILE *File_orbit = fopen("singleP_Evol.dat","w");  // Creates a File
			if(File_orbit == NULL){
				printf("Error File_orbit");
			exit(1);}  
			
			for(int t_=0;t_<(R_size);t_++){
				fprintf(File_orbit,"%f \t %f \t %f \n",R[t_].r[0], R[t_].r[1], R[t_].r[2]);
			}
			fclose(File_orbit);
			//D_HE = &He[ip];
			//printf("D_HE.r[0]=%f\n", (*D_HE).r[0]);  // Lo copia bien al puntero
			//Evol_w_coordinates (struct Part * d_He, long init, double &r_cor, double &theta_cor, double &z_cor)
			//singleP_Evol(D_HE, ip);
		}

		Part_charge[He[ip].q] = Part_charge[He[ip].q]+1;
		r = sqrt( He[ip].r[0]*He[ip].r[0] + He[ip].r[1]*He[ip].r[1]);
		x = sqrt( (r - hR0)*(r - hR0) + He[ip].r[2]*He[ip].r[2] );
		//printf("state= %e\n", He[ip].state);

		fprintf(File_FC," %d %f \t  %.5e \t %.5e \t %.5e \t%.5e \t %.5e \t %.5e \t %.5e \t %.5e \t %.5e \t %d  \n",
			//			He[ip].time, rg[0],rg[1],rg[2],
				ip,He[ip].time,He[ip].r[0], He[ip].r[1], He[ip].r[2],
				He[ip].v[0], He[ip].v[1], He[ip].v[2], 
				He[ip].E_keV,He[ip].flux,He[ip].pitch, He[ip].sense);

		if(He[ip].flag == 1){
			reentrantes++;
		}

		if(He[ip].state == 0){
			escapadas += 1;
		} else if(He[ip].state == 1){
			bananas += 1;
		} else if(He[ip].state == 2){
			clockW += 1;
		} else if(He[ip].state == 3){
			anticlockW += 1;
		} else if(He[ip].state == 4){
			Outliers += 1;
		} 
	}
	printf("%d, escaparon y volvieron!\n", reentrantes);
	fprintf(File_Orbit_types, "%d\t%d\t%d\t%d\t%d\n", escapadas, bananas, clockW, anticlockW, Outliers);


	//---------------------------------------

	gettimeofday(&finish,NULL);
	
	/* *** More statistical results ********** */
	fprintf(File_St,"\n Final \n");
	elapsed_time = (finish.tv_sec - start.tv_sec) + (finish.tv_usec - start.tv_usec)/1.0e6;
	fprintf(File_St,"Elapsed time: \t %f sec.\n", elapsed_time);
	//fprintf(File_Stat,"N process: \t %d \n", Nprocess);
	fprintf(File_St,"N He0: \t %d \n", Part_charge[0]);
	fprintf(File_St,"N He1: \t %d \n", Part_charge[1]);
	fprintf(File_St,"N He2: \t %d \n", Part_charge[2]);
	printf("Elapsed time: \t %f sec.\n", elapsed_time);
	
	fclose(File_IC);
	fclose(File_FC);
	fclose(File_St);
	fclose(File_Orbit_types);
	
	return 0;
}  /* end main */
